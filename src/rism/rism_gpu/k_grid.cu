#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism3d_grid.hpp"
using namespace std;

namespace rism3d_c {

    /////////////// KERNELS ///////////////

    __global__ void k_setup_wavevector(int n, int rx, int ry, int rz,
                                       GPUtype *wavevectorX, GPUtype *wavevectorY, GPUtype *wavevectorZ,
                                       GPUtype fatbLx, GPUtype fatbLy, GPUtype fatbLz,
                                       GPUtype *waveVectors2){
        int lgz, lgy, lgx, igk, igk2;
        GPUtype waveX, waveY, waveZ;
        int indX = blockIdx.x * blockDim.x + threadIdx.x;
        int indY = blockIdx.y * blockDim.y + threadIdx.y;
        int indZ = blockIdx.z * blockDim.z + threadIdx.z;

        // old column major version
        // if(indX >= rx/2 || indY >= ry || indZ >= rz){
        //     return;
        // }

        // new row major version
        // switching from rx/2,ry,rz to rx,ry,rz/2 is exchanging the values 
        // for WavevectorX and wavevectorZ while comparing to Fortran version.
        // This means we are considering different subvolumes in Fortran and
        // C++ implementations while working in the reciprocal space. So, results
        // of the direct Fourier transforms cannot be directly compared.
        // The wavenumber values does not change, though
        if(indX >= rx || indY >= ry || indZ >= rz/2){
            return;
        }

        // old column major version
        // igk = indX + (indY + indZ * ry) * rx / 2;

        // new row major version
        igk = indZ + (indY + indX * ry) * rz / 2;
    
        lgz = ((indZ + (rz / 2 - 1)) % rz) - (rz / 2 - 1);
        lgy = ((indY + (ry / 2 - 1)) % ry) - (ry / 2 - 1);
        lgx = ((indX + (rx / 2 - 1)) % rx) - (rx / 2 - 1);

        waveX = fatbLx*lgx;
        waveY = fatbLy*lgy;
        waveZ = fatbLz*lgz;

        wavevectorX[igk] = waveX;
        wavevectorY[igk] = waveY;
        wavevectorZ[igk] = waveZ;

        waveVectors2[igk] = waveX*waveX + waveY*waveY + waveZ*waveZ;

        // Nyquist frequencies

        // old column major version
        // igk2 = indY + (indZ + rz * rx/2) * ry;

        // new row major version
        igk2 = indY + (indX + rx * rz/2) * ry;

        // old column major version
        // GPUtype waveX2;
        // int lgx2;
        // lgx2 = ((rx / 2 + (rx / 2 - 1)) % rx) - (rx / 2 - 1);
        // waveX2 = fatbLx*lgx2;

        // new row major version
        GPUtype waveZ2;
        int lgz2;
        lgz2 = ((rz / 2 + (rz / 2 - 1)) % rz) - (rz / 2 - 1);
        waveZ2 = fatbLz*lgz2;

        wavevectorX[igk2] = waveX;
        wavevectorY[igk2] = waveY;
        wavevectorZ[igk2] = waveZ2;

        // old column major version
        // waveVectors2[igk2] = waveX2*waveX2 + waveY*waveY + waveZ*waveZ;

        // new row major version
        waveVectors2[igk2] = waveX*waveX + waveY*waveY + waveZ2*waveZ2;

    }

    /////////////// KERNEL WRAPPER FUNCTIONS ///////////////

    void rism3d_grid :: setup_wavevector(){
        GPUtype pi;
        GPUtype factor[3];

        // Defining and allocating memory for waveVectorToWaveVector2Map
        array_class<int> waveVectorToWaveVector2Map;
        waveVectorToWaveVector2Map.set_memalloc(memalloc_p);
        waveVectorToWaveVector2Map.alloc_mem(totalLocalPointsK/2);

        // allocate memory for waveVectorWaveNumberMap
        waveVectorWaveNumberMap.set_memalloc(memalloc_p, true);
        waveVectorWaveNumberMap.alloc_mem(totalLocalPointsK/2);

#if RISMCUDA_DOUBLE
        pi = M_PI;
#else
        pi = (float)M_PI;
#endif // RISMCUDA_DOUBLE

        waveVectorX.set_memalloc(memalloc_p, true);
        waveVectorX.alloc_mem(totalLocalPointsK/2);
        hipMemPrefetchAsync(waveVectorX.m_data, (totalLocalPointsK/2)*sizeof(GPUtype), 0);

        waveVectorY.set_memalloc(memalloc_p, true);
        waveVectorY.alloc_mem(totalLocalPointsK/2);
        hipMemPrefetchAsync(waveVectorY.m_data, (totalLocalPointsK/2)*sizeof(GPUtype), 0);

        waveVectorZ.set_memalloc(memalloc_p, true);
        waveVectorZ.alloc_mem(totalLocalPointsK/2);
        hipMemPrefetchAsync(waveVectorZ.m_data, (totalLocalPointsK/2)*sizeof(GPUtype), 0);

        waveVectors2.set_memalloc(memalloc_p, true);
        waveVectors2.alloc_mem(totalLocalPointsK/2);
        hipMemPrefetchAsync(waveVectors2.m_data, (totalLocalPointsK/2)*sizeof(GPUtype), 0);
        hipDeviceSynchronize();
        hipMemset(waveVectors2.m_data, 0, totalLocalPointsK/2 * sizeof(GPUtype));

        // ?? Do we need a synchronization step here?

        // The number of blocks in each thread dimention may need to be adjusted for optimization
        int n_blocksx = 8;
        int n_blocksy = 8;
        int n_blocksz = 8;
        dim3 block(n_blocksx,n_blocksy,n_blocksz);
        dim3 grid((globalDimsR[0] + n_blocksx-1) / n_blocksx, (globalDimsR[1] + n_blocksy-1) / n_blocksy, (globalDimsR[2] + n_blocksz-1) / n_blocksz);

        for(int i = 0; i < 3; i++){
            factor[i] = (2.0 * pi) / boxLength[i];
        }

        k_setup_wavevector<<<grid,block>>>(totalLocalPointsK/2,
                                           globalDimsR[0],globalDimsR[1],globalDimsR[2],
                                           waveVectorX.m_data, waveVectorY.m_data, waveVectorZ.m_data,
                                           factor[0],factor[1],factor[2], waveVectors2.m_data);
        hipDeviceSynchronize();

        // Getting waveVectors2 data back to CPU
        hipMemPrefetchAsync(waveVectors2.m_data, (totalLocalPointsK/2)*sizeof(GPUtype), hipCpuDeviceId);

        // waveVectorToWaveVector2Map is an index array that will retrieve waveVectors2 in 
        // ascending order
        // Ideally, this indexArray function would be implemented as a CUDA kernel. However,
        // I think it is ok to have this as a serial implementation because maybe the speed 
        // up would not be significant. Also, the siftdown function may not be trivial to
        // parallelize
        indexArray(waveVectors2.m_data, waveVectorToWaveVector2Map.m_data, totalLocalPointsK/2);

        // print test for waveVecotrToWaveVector2Map
        // Comparing to Fortran results may have some different indexes because of repeated
        // values (e.g. 0.026127333114358848) happening more than once. Thus, some indexes 
        // refering to the same value can appear with switched positions in Fortran.
        // This print statement shows that using waveVectorToWaveVector2Map ensures
        // waveVectors2 will be get in ascending order
        // for(int i = 0; i < 20; i++){
        //     cout << "waveVectors2[" << " waveVectorToWaveVector2Map[" << i << "]] = " << waveVectors2(waveVectorToWaveVector2Map(i)) << endl;
        // }
        // for(int i = (totalLocalPointsK/2)-10; i < totalLocalPointsK/2; i++){
        //     cout << "waveVectors2[" << " waveVectorToWaveVector2Map[" << i << "]] = " << waveVectors2(waveVectorToWaveVector2Map(i)) << endl;
        // }

        waveNumbers.set_memalloc(memalloc_p, true);

        // Ideally, this sort_unique function would be implemented as a CUDA kernel,
        // since it is working with waveNumbers and waveVectorWaveNumberMap.
        // However, I think we can left it for later because:
        //     1 - It may not be that trivial to parallelize it
        //     2 - I do not think we would get a significant speed up
        //     3 - I think the slow down of sending data from CPU to GPU in this case
        //         would not be significant, because would be done only once.
        sort_unique(&waveVectors2, totalLocalPointsK/2, &waveNumbers, &waveVectorToWaveVector2Map, &waveVectorWaveNumberMap);

        // print test for waveVectorWaveNumberMap
        // Here we can see that waveNumbers(waveVectorWaveNumberMap(i)) will  return the correct value of
        // sqrt(waveVectors2(i)). Thus, waveVectorWaveNumberMap(i) maps the indexing correctly.
        // for(int i = 0; i < 20; i++){
        //     cout << "waveVectors2[" << i << "] = " << waveVectors2(i) << " || waveNumber = " << waveNumbers(waveVectorWaveNumberMap(i)) << endl;
        // }
        // for(int i = (totalLocalPointsK/2)-10; i < totalLocalPointsK/2; i++){
        //     cout << "waveVectors2[" << i << "] = " << waveVectors2(i) << " || waveNumber = " << waveNumbers(waveVectorWaveNumberMap(i)) << endl;
        // }

        waveNumberArraySize = waveNumbers.get_dim1();
        
    }

}