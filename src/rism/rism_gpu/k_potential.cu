#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism3d_potential.hpp"
using namespace std;

namespace rism3d_c {

    // Define PI as a constant on device
    __device__ const GPUtype PI = 3.14159265358979323846;

#if RISMCUDA_DOUBLE
    __device__ const GPUtype max_value = DBL_MAX;
#else
    __device__ const GPUtype max_value = FLT_MAX;
#endif // RISMCUDA_DOUBLE

    // Kernel to calculate Lennard-Jones potential
    __global__ void k_potential_calc(GPUtype *lj, GPUtype *A, GPUtype *B, 
                                     int solvent_numAtomTypes, int solute_numAtoms, 
                                     int x_dim, int y_dim, int z_dim, 
                                     GPUtype *pos,
                                     GPUtype gridspc_x, GPUtype gridspc_y, GPUtype gridspc_z){
        GPUtype rx,ry,rz;
        GPUtype dz2,dy2,dx2;
        GPUtype rs2i, rs6i;
        GPUtype r2;

        GPUPotAccumType potential = 0.0;

        // Use long long int to handle large systems
        long long int idx = threadIdx.x + (long long int)blockIdx.x * blockDim.x;

        if (idx < (long long int)solvent_numAtomTypes * x_dim * y_dim * z_dim) {
            int i = idx / (x_dim * y_dim * z_dim); // solvent numAtomTypes index
            int m = (idx / (y_dim * z_dim)) % x_dim; // x index
            int l = (idx / z_dim) % y_dim; // y index
            int k = idx % z_dim; // z index
            for(int solu_n = 0; solu_n < solute_numAtoms; solu_n++){

                int j = solu_n; // solute numAtoms idex
    
                // Checking out-of-bound access
                // int mem_idx = i*x_dim*y_dim*(z_dim + 2) + m*y_dim*(z_dim + 2) + l*(z_dim + 2) + k;
                // int max_size = solvent_numAtomTypes * x_dim * y_dim * (z_dim + 2);
                // if (mem_idx >= max_size) {
                //     printf("Thread %lld attempted out-of-bounds access: %d >= %d\n", idx, mem_idx, max_size);
                // }

                rz = k*gridspc_z;
                // If pos is column major
                // dz2 = (rz - pos[2 + j*3])*(rz - pos[2 + j*3]);
                // If pos is row major
                dz2 = (rz - pos[2*solute_numAtoms + j])*(rz - pos[2*solute_numAtoms + j]);
                
                ry = l*gridspc_y;
                // If pos is column major
                // dy2 = (ry - pos[1 + j*3])*(ry - pos[1 + j*3]);
                // If pos is row major
                dy2 = (ry - pos[1*solute_numAtoms + j])*(ry - pos[1*solute_numAtoms + j]);

                rx = m*gridspc_x;
                // If pos is column major
                // dx2 = (rx - pos[0 + j*3])*(rx - pos[0 + j*3]);
                // If pos is row major
                dx2 = (rx - pos[0*solute_numAtoms + j])*(rx - pos[0*solute_numAtoms + j]);

                r2 = dx2 + dy2 + dz2;

                rs2i = (GPUtype)1.0/r2;
                rs6i = rs2i*rs2i*rs2i;

                potential += rs6i*(rs6i*A[j*solvent_numAtomTypes+i] - B[j*solvent_numAtomTypes+i]);

            }
            // old column major order
            // lj[m + l*x_dim + k*x_dim*y_dim + i*x_dim*y_dim*z_dim] += rs6i*(rs6i*A[j*solvent_numAtomTypes+i] - B[j*solvent_numAtomTypes+i]);

            // add 2 to z_dim in order to account the extra padding not used
            lj[i*x_dim*y_dim*(z_dim + 2) + m*y_dim*(z_dim + 2) + l*(z_dim + 2) + k] += static_cast<GPUtype>(potential);

            if(lj[i*x_dim*y_dim*(z_dim + 2) + m*y_dim*(z_dim + 2) + l*(z_dim + 2) + k] > max_value || isnan(lj[i*x_dim*y_dim*(z_dim + 2) + m*y_dim*(z_dim + 2) + l*(z_dim + 2) + k])){
                lj[i*x_dim*y_dim*(z_dim + 2) + m*y_dim*(z_dim + 2) + l*(z_dim + 2) + k] = sqrt(max_value);
            }

        }
    }

     // Kernel to calculate Coulomb potential
     __global__ void k_coulomb_potential_calc(int target_x_low_ind,  int target_x_high_ind,
        int target_y_low_ind,  int target_y_high_ind,
        int target_z_low_ind,  int target_z_high_ind,
        GPUtype grid_xmin,      GPUtype grid_ymin,      GPUtype grid_zmin,
        GPUtype grid_spacing_x,       GPUtype grid_spacing_y,       GPUtype grid_spacing_z,
        int grid_dim_x,   int grid_dim_y,   int grid_dim_z,
        int solute_numAtoms, int solute_numAtoms_idx_start,
        const GPUtype *solute_position_x, const GPUtype *solute_position_y, const GPUtype *solute_position_z, const GPUtype *solute_charge,
        GPUtype *potential, GPUtype solvent_charge){

        // Remeber: our potential array has (two) extra paddings in the z dimentions
        // Thus, we need to account this to get the correct indexes
        int target_yz_dim = grid_dim_y * (grid_dim_z + 2);

        // Compute 3D thread and block indices
        int ix = blockIdx.x * blockDim.x + threadIdx.x + target_x_low_ind;
        int iy = blockIdx.y * blockDim.y + threadIdx.y + target_y_low_ind;
        int iz = blockIdx.z * blockDim.z + threadIdx.z + target_z_low_ind;

        if (ix > target_x_high_ind || iy > target_y_high_ind || iz > target_z_high_ind) return;

        int ii = (ix * target_yz_dim) + (iy * (grid_dim_z + 2)) + iz;
        GPUPotAccumType temporary_potential = 0.0;

        GPUtype tx = grid_xmin + (ix - target_x_low_ind) * grid_spacing_x;
        GPUtype ty = grid_ymin + (iy - target_y_low_ind) * grid_spacing_y;
        GPUtype tz = grid_zmin + (iz - target_z_low_ind) * grid_spacing_z;

        for (int j = 0; j < solute_numAtoms; j++) {
            int jj = solute_numAtoms_idx_start + j;

            GPUtype dx = tx - solute_position_x[jj];
            GPUtype dy = ty - solute_position_y[jj];
            GPUtype dz = tz - solute_position_z[jj];
            GPUtype r  = sqrt(dx*dx + dy*dy + dz*dz);

            if (r > 0) {
            temporary_potential += static_cast<GPUPotAccumType>(solute_charge[jj] / r);
            }
        }

        // Use atomicAdd to safely update the shared potential array
        // atomicAdd(&potential[ii], temporary_potential);
        // No atomicAdd needed, since each thread handles a unique ii
        potential[ii] = static_cast<GPUtype>(temporary_potential * solvent_charge);

        if(potential[ii] > max_value || isnan(potential[ii])){
            potential[ii] = sqrt(max_value);
        }

    }

    void rism3d_potential :: potential_calc(){
        // TO DO: Check if grid size was changed
        
        // Allocating memory using globalDimsK to account the extra padding necessary for huv array
        uuv.alloc_mem(solventclass_p->numAtomTypes, grid_p->globalDimsK[0], grid_p->globalDimsK[1], grid_p->globalDimsK[2]);
        hipMemset(uuv.m_data, 0, grid_p->globalDimsK[0]*grid_p->globalDimsK[1]*grid_p->globalDimsK[2]*solventclass_p->numAtomTypes*sizeof(GPUtype));

        // Prefetching arrays that will be used on the device functions
        int device = -1;
        hipGetDevice(&device);
        hipMemPrefetchAsync(ljAUV.m_data, solventclass_p->numAtomTypes * soluteclass_p->numAtoms * sizeof(GPUtype), device, NULL);
        hipMemPrefetchAsync(ljBUV.m_data, solventclass_p->numAtomTypes * soluteclass_p->numAtoms * sizeof(GPUtype), device, NULL);
        hipMemPrefetchAsync(soluteclass_p->position.m_data, 3 * soluteclass_p->numAtoms * sizeof(GPUtype), device, NULL);
        hipMemPrefetchAsync(uuv.m_data, solventclass_p->numAtomTypes*grid_p->globalDimsK[0]*grid_p->globalDimsK[1]*grid_p->globalDimsK[2] * sizeof(GPUtype), device, NULL);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            cout << "Probable error with hipMemPrefetchAsync" << hipGetErrorString(err) << endl;
            abort();
        }

        if(soluteclass_p->charged){
            if(periodic == true){
                cout << "Sorry: periodic systems not supported, yet!" << endl;
                abort();
            }
            else{
                if(treeCoulomb == true){
                    cout << "tree code version not available, yet." << endl;
                    cout << "Run using --notreeCoulomb flag" << endl;
                    abort();
                }
                else{
                    dim3 blockDim(8, 8, 8);
                    dim3 gridDim((grid_p->localDimsR[0] + blockDim.x) / blockDim.x,
                                 (grid_p->localDimsR[1] + blockDim.y) / blockDim.y,
                                 (grid_p->localDimsR[2] + blockDim.z) / blockDim.z);


                    // Iterate over solvent atom types to run calculations simultaneously
                    // and synchronize only after the for loop
                    for(int iv = 0; iv < solventclass_p->numAtomTypes; iv++){
                        k_coulomb_potential_calc<<<gridDim, blockDim>>>(0, grid_p->localDimsR[0] - 1,
                                                                        0, grid_p->localDimsR[1] - 1,
                                                                        0, grid_p->localDimsR[2] - 1,
                                                                        0, 0, 0,
                                                                        grid_p->spacing[0], grid_p->spacing[1], grid_p->spacing[2],
                                                                        grid_p->localDimsR[0], grid_p->localDimsR[1], grid_p->localDimsR[2],
                                                                        soluteclass_p->numAtoms, 0,
                                                                        soluteclass_p->position.m_data, 
                                                                        soluteclass_p->position.m_data + soluteclass_p->numAtoms, 
                                                                        soluteclass_p->position.m_data + 2*soluteclass_p->numAtoms, 
                                                                        soluteclass_p->charge.m_data,
                                                                        uuv.m_data + iv * grid_p->localDimsR[0] * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2),
                                                                        solventclass_p->charge.m_data[iv]);
                    }
                    hipError_t err2 = hipGetLastError();
                    if (err2 != hipSuccess) {
                        cout << "k_coulomb_potential_calc kernel launch failed: " << hipGetErrorString(err2) << endl;
                        abort();
                    }
                    hipDeviceSynchronize();

                    // Saving values to compare with fortran version: i am keeping this for now
// #if RISMCUDA_DOUBLE
//                     ofstream file1("/home/fcarvalho/rism3d.cuda.test.chg/coulomb_1_db.txt");
//                     ofstream file2("/home/fcarvalho/rism3d.cuda.test.chg/coulomb_2_db.txt");
// #else
//                     ofstream file1("/home/fcarvalho/rism3d.cuda.test.chg/coulomb_1_float.txt");
//                     ofstream file2("/home/fcarvalho/rism3d.cuda.test.chg/coulomb_2_float.txt");
// #endif // RISMCUDA_DOUBLE
//                     file1 << std::scientific << std::setprecision(16);
//                     file2 << std::scientific << std::setprecision(16);
//                     for(int ix = 0; ix < grid_p->localDimsR[0]; ix++){
//                         for(int iy = 0; iy < grid_p->localDimsR[1]; iy++){
//                             for(int iz = 0; iz < grid_p->localDimsR[2]; iz++){
//                                 file1 << uuv.m_data[0 * grid_p->localDimsR[0] * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                                     ix * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                                     iy * (grid_p->localDimsR[2] + 2) + 
//                                                     iz] << endl;
//                                 file2 << uuv.m_data[1 * grid_p->localDimsR[0] * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                                     ix * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                                     iy * (grid_p->localDimsR[2] + 2) + 
//                                                     iz] << endl;
//                             }
//                         }
//                     }
//                     file1.close();
//                     file2.close();
                }
            }
        }

        if(periodic == true){
            cout << "Sorry: periodic systems not supported, yet!" << endl;
            abort();
        } else{
            // using long long type to ensure we will not get overflow while computing num_blocks
            long long num_blocks = (static_cast<long long>(solventclass_p->numAtomTypes) * static_cast<long long>(soluteclass_p->numAtoms) * 
                                    static_cast<long long>(grid_p->globalDimsR[0]) * static_cast<long long>(grid_p->globalDimsR[1]) * 
                                    static_cast<long long>(grid_p->globalDimsR[2]) + 255) / 256;
            int num_threads = 256;

            // Checking GPU kernel properties (uncomment below to see values)
            // hipDeviceProp_t prop;
            // hipGetDeviceProperties(&prop, 0);
            // cout << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << endl;

            k_potential_calc<<<num_blocks, num_threads>>>(uuv.m_data, ljAUV.m_data, ljBUV.m_data, 
                                                          solventclass_p->numAtomTypes, soluteclass_p->numAtoms, 
                                                          grid_p->globalDimsR[0], grid_p->globalDimsR[1], grid_p->globalDimsR[2], 
                                                          soluteclass_p->position.m_data,
                                                          grid_p->spacing[0], grid_p->spacing[1], grid_p->spacing[2]);
            
            hipDeviceSynchronize();
            hipError_t err3 = hipGetLastError();
            if (err3 != hipSuccess) {
                cout << "k_potential_calc kernel launch failed: " << hipGetErrorString(err3) << endl;
                abort();
            }

        }

        // Saving values for full potential to compare with Fortran version: i am keeping this for now
// #if RISMCUDA_DOUBLE
//         ofstream file1("/home/fcarvalho/rism3d.cuda.test.chg/full_pot_1_db.txt");
//         ofstream file2("/home/fcarvalho/rism3d.cuda.test.chg/full_pot_2_db.txt");
// #else
//         ofstream file1("/home/fcarvalho/rism3d.cuda.test.chg/full_pot_1_float.txt");
//         ofstream file2("/home/fcarvalho/rism3d.cuda.test.chg/full_pot_2_float.txt");
// #endif // RISMCUDA_DOUBLE
//         file1 << std::scientific << std::setprecision(16);
//         file2 << std::scientific << std::setprecision(16);
//         for(int ix = 0; ix < grid_p->localDimsR[0]; ix++){
//             for(int iy = 0; iy < grid_p->localDimsR[1]; iy++){
//                 for(int iz = 0; iz < grid_p->localDimsR[2]; iz++){
//                     file1 << uuv.m_data[0 * grid_p->localDimsR[0] * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                         ix * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                         iy * (grid_p->localDimsR[2] + 2) + 
//                                         iz] << endl;
//                     file2 << uuv.m_data[1 * grid_p->localDimsR[0] * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                         ix * grid_p->localDimsR[1] * (grid_p->localDimsR[2] + 2) + 
//                                         iy * (grid_p->localDimsR[2] + 2) + 
//                                         iz] << endl;
//                 }
//             }
//         }
//         file1.close();
//         file2.close();
        
    }

    __global__ void K_dcf_long_range_asymptotics_R(int target_x_low_ind,    int target_x_high_ind,
                                                   int target_y_low_ind,    int target_y_high_ind,
                                                   int target_z_low_ind,    int target_z_high_ind,
                                                   GPUtype grid_xmin,      GPUtype grid_ymin,      GPUtype grid_zmin,
                                                   GPUtype grid_spacing_x,       GPUtype grid_spacing_y,       GPUtype grid_spacing_z,
                                                   int grid_dim_x,   int grid_dim_y,   int grid_dim_z,
                                                   int solute_numAtoms, int solute_numAtoms_idx_start,
                                                   const GPUtype *solute_position_x, const GPUtype *solute_position_y, const GPUtype *solute_position_z, const GPUtype *solute_charge,
                                                   GPUtype eta, GPUtype *dcf_long_range_asymptotics)
    {
        // Compute global indices for this thread
        int ix = blockIdx.x * blockDim.x + threadIdx.x + target_x_low_ind;
        int iy = blockIdx.y * blockDim.y + threadIdx.y + target_y_low_ind;
        int iz = blockIdx.z * blockDim.z + threadIdx.z + target_z_low_ind;

        // Check bounds
        if (ix > target_x_high_ind || iy > target_y_high_ind || iz > target_z_high_ind) {
            return;
        }

        int target_yz_dim = grid_dim_y * grid_dim_z;
        int ii = (ix * target_yz_dim) + (iy * grid_dim_z) + iz;

        GPUtype tx = grid_xmin + (ix - target_x_low_ind) * grid_spacing_x;
        GPUtype ty = grid_ymin + (iy - target_y_low_ind) * grid_spacing_y;
        GPUtype tz = grid_zmin + (iz - target_z_low_ind) * grid_spacing_z;

        GPUPotAccumType temporary_dcf_long_range_asymptotics = 0.0;

        // Each thread processes all source points for one (ix, iy, iz)
        for (int j = 0; j < solute_numAtoms; j++) {
            int jj = solute_numAtoms_idx_start + j;
            GPUtype dx = tx - solute_position_x[jj];
            GPUtype dy = ty - solute_position_y[jj];
            GPUtype dz = tz - solute_position_z[jj];
            GPUtype r  = sqrt(dx*dx + dy*dy + dz*dz);

            if (r > 0) {
                temporary_dcf_long_range_asymptotics -= static_cast<GPUPotAccumType>(solute_charge[jj] * erf(r / eta) / r);
            }
            else{
                temporary_dcf_long_range_asymptotics -= static_cast<GPUPotAccumType>(solute_charge[jj] / (sqrt(PI) * eta) * 2.0);
            }
        }

        // No atomicAdd needed, since each thread handles a unique ii
        dcf_long_range_asymptotics[ii] = static_cast<GPUtype>(temporary_dcf_long_range_asymptotics);
    }

    void rism3d_potential :: dcf_long_range_asymptotics_R(int target_x_low_ind, int target_x_high_ind,
                                                          int target_y_low_ind, int target_y_high_ind,
                                                          int target_z_low_ind, int target_z_high_ind,
                                                          GPUtype grid_xmin, GPUtype grid_ymin, GPUtype grid_zmin,
                                                          GPUtype grid_spacing_x, GPUtype grid_spacing_y, GPUtype grid_spacing_z,
                                                          int grid_dim_x, int grid_dim_y, int grid_dim_z,
                                                          int solute_numAtoms, int solute_numAtoms_idx_start,  
                                                          GPUtype *solute_position_x, GPUtype *solute_position_y, GPUtype *solute_position_z, GPUtype *solute_charge,
                                                          GPUtype eta, GPUtype *dcf_long_range_asymptotics){
    
        // Compute grid and block dimensions
        int x_range = target_x_high_ind - target_x_low_ind + 1;
        int y_range = target_y_high_ind - target_y_low_ind + 1;
        int z_range = target_z_high_ind - target_z_low_ind + 1;

        // Choose block dimensions (tune as necessary)
        dim3 blockDim(8, 8, 8);
        dim3 gridDim((x_range + blockDim.x - 1) / blockDim.x,
                     (y_range + blockDim.y - 1) / blockDim.y,
                     (z_range + blockDim.z - 1) / blockDim.z);

        // Choose a CUDA stream if gpu_async_stream_id corresponds to a valid hipStream_t.
        // Here we assume you have a pre-created hipStream_t array or handle mapping to gpu_async_stream_id.
        // If not using streams, just use the default stream (0).
        hipStream_t stream = 0; // Replace with appropriate stream handle if available

        // Launch the CUDA kernel
        K_dcf_long_range_asymptotics_R<<<gridDim, blockDim, 0, stream>>>(target_x_low_ind, target_x_high_ind,
                                                                         target_y_low_ind, target_y_high_ind,
                                                                         target_z_low_ind, target_z_high_ind,
                                                                         grid_xmin, grid_ymin, grid_zmin,
                                                                         grid_spacing_x, grid_spacing_y, grid_spacing_z,
                                                                         grid_dim_x, grid_dim_y, grid_dim_z,
                                                                         solute_numAtoms, solute_numAtoms_idx_start,
                                                                         solute_position_x, solute_position_y, solute_position_z, solute_charge,
                                                                         eta, dcf_long_range_asymptotics);

        // Wait for the kernel to finish here
        // hipStreamSynchronize(stream);
        hipDeviceSynchronize();
    }

    __global__ void K_tcf_long_range_asymptotics_R(int target_x_low_ind, int target_x_high_ind,
                                                   int target_y_low_ind, int target_y_high_ind,
                                                   int target_z_low_ind, int target_z_high_ind,
                                                   GPUtype grid_xmin, GPUtype grid_ymin, GPUtype grid_zmin,
                                                   GPUtype grid_spacing_x, GPUtype grid_spacing_y, GPUtype grid_spacing_z,
                                                   int grid_dim_x, int grid_dim_y, int grid_dim_z,
                                                   int solute_numAtoms, int solute_numAtoms_idx_start,  
                                                   GPUtype *solute_position_x, GPUtype *solute_position_y, GPUtype *solute_position_z, GPUtype *solute_charge,
                                                   GPUtype kap, GPUtype eta, GPUtype solvent_dielconst, GPUtype *tcf_long_range_asymptotics){
        
        // Calculate 3D grid indices
        int ix = blockIdx.x * blockDim.x + threadIdx.x + target_x_low_ind;
        int iy = blockIdx.y * blockDim.y + threadIdx.y + target_y_low_ind;
        int iz = blockIdx.z * blockDim.z + threadIdx.z + target_z_low_ind;

        if (ix > target_x_high_ind || iy > target_y_high_ind || iz > target_z_high_ind)
            return; // Out-of-bounds check

        int target_yz_dim = grid_dim_y * grid_dim_z;
        GPUtype kap_eta_2 = kap * eta / 2.0;

        // Compute target index in potential array
        int ii = (ix * target_yz_dim) + (iy * grid_dim_z) + iz;

        // Compute target coordinates
        GPUtype tx = grid_xmin + (ix - target_x_low_ind) * grid_spacing_x;
        GPUtype ty = grid_ymin + (iy - target_y_low_ind) * grid_spacing_y;
        GPUtype tz = grid_zmin + (iz - target_z_low_ind) * grid_spacing_z;

        GPUPotAccumType temporary_tcf_long_range_asymptotics = 0.0;

        // Loop over sources
        for (int j = 0; j < solute_numAtoms; j++) {
            int jj = solute_numAtoms_idx_start + j;
            GPUtype dx = tx - solute_position_x[jj];
            GPUtype dy = ty - solute_position_y[jj];
            GPUtype dz = tz - solute_position_z[jj];
            GPUtype r  = sqrt(dx * dx + dy * dy + dz * dz);

            GPUtype kap_r = kap * r;
            GPUtype r_eta = r / eta;

            if (r > 0) {
                temporary_tcf_long_range_asymptotics += static_cast<GPUPotAccumType>(- solute_charge[jj] / r
                                                        * (exp(-kap_r) * erfc(kap_eta_2 - r_eta)
                                                        -  exp( kap_r) * erfc(kap_eta_2 + r_eta)) / 2);
            }
            else{
                temporary_tcf_long_range_asymptotics += static_cast<GPUPotAccumType>(- solute_charge[jj]
                                                        * (2 / (sqrt(PI) * eta)
                                                        -  exp( kap_eta_2 * kap_eta_2) * kap * erfc(kap_eta_2)) / exp( kap_eta_2 * kap_eta_2));
                }
        }

        // Atomic update to prevent race conditions
        // atmicAdd(&potential[ii], temporary_tcf_long_range_asymptotics);
        // No atomicAdd needed, since each thread handles a unique ii
        tcf_long_range_asymptotics[ii] = static_cast<GPUtype>(temporary_tcf_long_range_asymptotics * exp(kap_eta_2 * kap_eta_2) / solvent_dielconst);
    }

    void rism3d_potential :: tcf_long_range_asymptotics_R(int target_x_low_ind, int target_x_high_ind,
                                                          int target_y_low_ind, int target_y_high_ind,
                                                          int target_z_low_ind, int target_z_high_ind,
                                                          GPUtype grid_xmin, GPUtype grid_ymin, GPUtype grid_zmin,
                                                          GPUtype grid_spacing_x, GPUtype grid_spacing_y, GPUtype grid_spacing_z,
                                                          int grid_dim_x, int grid_dim_y, int grid_dim_z,
                                                          int solute_numAtoms, int solute_numAtoms_idx_start,  
                                                          GPUtype *solute_position_x, GPUtype *solute_position_y, GPUtype *solute_position_z, GPUtype *solute_charge,
                                                          GPUtype xappa, GPUtype chargeSmear, GPUtype solvent_dielconst, GPUtype *tcf_long_range_asymptotics){

        // Compute grid and block dimensions
        int x_range = target_x_high_ind - target_x_low_ind + 1;
        int y_range = target_y_high_ind - target_y_low_ind + 1;
        int z_range = target_z_high_ind - target_z_low_ind + 1;

        // Choose block dimensions (tune as necessary)
        dim3 blockDim(8, 8, 8);
        dim3 gridDim((x_range + blockDim.x - 1) / blockDim.x,
                     (y_range + blockDim.y - 1) / blockDim.y,
                     (z_range + blockDim.z - 1) / blockDim.z);

        // Choose a CUDA stream if gpu_async_stream_id corresponds to a valid hipStream_t.
        // Here we assume you have a pre-created hipStream_t array or handle mapping to gpu_async_stream_id.
        // If not using streams, just use the default stream (0).
        hipStream_t stream = 0; // Replace with appropriate stream handle if available

        // Launch the CUDA kernel
        K_tcf_long_range_asymptotics_R<<<gridDim, blockDim, 0, stream>>>(target_x_low_ind, target_x_high_ind,
                                                                         target_y_low_ind, target_y_high_ind,
                                                                         target_z_low_ind, target_z_high_ind,
                                                                         grid_xmin, grid_ymin, grid_zmin,
                                                                         grid_spacing_x, grid_spacing_y, grid_spacing_z,
                                                                         grid_dim_x, grid_dim_y, grid_dim_z,
                                                                         solute_numAtoms, solute_numAtoms_idx_start,
                                                                         solute_position_x, solute_position_y, solute_position_z, solute_charge,
                                                                         xappa, chargeSmear, solvent_dielconst, tcf_long_range_asymptotics);

        // Wait for the kernel to finish here
        // hipStreamSynchronize(stream);
        hipDeviceSynchronize();

    }

    __global__ void K_calc_dcf_tcf_LongRangeAsympK(GPUtype* waveVectorX, // [numWaveVectors]
                                     GPUtype* waveVectorY, // [numWaveVectors]
                                     GPUtype* waveVectorZ, // [numWaveVectors]
                                     GPUtype* waveVectors2,
                                     GPUtype cut2_chlk,
                                     GPUtype* position,    // [3 * numAtoms] (x, y, z for all atoms)
                                     GPUtype* charge,      // [numAtoms] (charges of the solute atoms)
                                     GPUtype* dcfLongRangeAsympK,
                                     GPUtype asympk_const,
                                     GPUtype smear2_4,
                                     int numAtoms,
                                     int numWaveVectors_2,
                                     int start_ind,
                                     bool ionic, 
                                     GPUtype* tcfLongRangeAsympK,
                                     GPUtype xappa2, 
                                     GPUtype solvent_dielconst)
    {
        int ig = blockIdx.x * blockDim.x + threadIdx.x + start_ind;
        if (ig >= numWaveVectors_2) return; // Out of bounds check

        if(waveVectors2[ig] > cut2_chlk){
            return;
        }
        else{
            // Extract wave vector components for this thread
            GPUtype kx = waveVectorX[ig];
            GPUtype ky = waveVectorY[ig];
            GPUtype kz = waveVectorZ[ig];

            GPUPotAccumType sumCos = 0;
            GPUPotAccumType sumSin = 0;

            for (int iu = 0; iu < numAtoms; ++iu) {
                // Compute indices for x, y, z components of the atom
                GPUtype x = position[iu];
                GPUtype y = position[iu + numAtoms];
                GPUtype z = position[iu + 2 * numAtoms];

                // Compute the phase: dot product of wave vector and atom position
                GPUtype phase = kx * x + ky * y + kz * z;

                // Accumulate cosine and sine contributions
                sumCos = sumCos + static_cast<GPUPotAccumType>(charge[iu] * cos(phase));

                // Because we are using half of z axis for the in-place FFTW 
                // our CUDA version needs to use the 
                // complex conjugate of exp(i*dot(k,R))
                // (i.e. using a minus sign for the sine function)
                // to get the correct result.
                // There is a phase shift compared to Fortran version,
                // which takes half of x axis for the FFTW.
                sumSin = sumSin - static_cast<GPUPotAccumType>(charge[iu] * sin(phase));
            }

            GPUtype uc1g = asympk_const * exp(-smear2_4 * waveVectors2[ig]);
            GPUtype uc1gc = uc1g / waveVectors2[ig];

            dcfLongRangeAsympK[2*ig] = static_cast<GPUtype>(uc1gc * sumCos);
            dcfLongRangeAsympK[2*ig+1] = static_cast<GPUtype>(uc1gc * sumSin);

            if(ionic == true){
                GPUtype uc1gh = uc1g / ((waveVectors2[ig] + xappa2) * solvent_dielconst);
                tcfLongRangeAsympK[2*ig] = static_cast<GPUtype>(uc1gh * sumCos);
                tcfLongRangeAsympK[2*ig+1] = static_cast<GPUtype>(uc1gh * sumSin);
            }


        }
    }

    // Serial version for debugging: I will keep it here for now
    void rism3d_potential :: computeSumCosSin_serial(GPUtype* waveVectorX,
                                          GPUtype* waveVectorY,
                                          GPUtype* waveVectorZ,
                                          GPUtype* waveVectors2,
                                          GPUtype cut2_chlk,
                                          GPUtype* position,
                                          GPUtype* charge,
                                          GPUtype* dcfLongRangeAsympK,
                                          GPUtype asympk_const,
                                          GPUtype smear2_4,
                                          int numAtoms,
                                          int numWaveVectors_2,
                                          int start_ind){

        int index = 0;
        for(int i = start_ind; i < numWaveVectors_2; i++){
            if(waveVectors2[i] < cut2_chlk){
                GPUtype sumCos = 0;
                GPUtype sumSin = 0;
                for(int j = 0; j < numAtoms; j++){
                    GPUtype phase = position[j]*waveVectorX[i] + position[j + numAtoms]*waveVectorY[i] + position[j + 2*numAtoms]*waveVectorZ[i];
                    sumCos = sumCos + charge[j] * cos(phase);
                    sumSin = sumSin + charge[j] * sin(phase);
                }
                if(index < 4){
                    cout << sumCos << endl;
                    cout << sumSin << endl;
                    index = index + 1;
                }
                GPUtype uc1g = asympk_const * exp(-smear2_4 * waveVectors2[i]);
                GPUtype uc1gc = uc1g / waveVectors2[i];
                dcfLongRangeAsympK[2*i] = uc1gc * sumCos;
                dcfLongRangeAsympK[2*i+1] = uc1gc * sumSin;
            }
        }

    }

    void rism3d_potential :: calc_dcf_tcf_LongRangeAsympK(GPUtype* waveVectorX,
                                          GPUtype* waveVectorY,
                                          GPUtype* waveVectorZ,
                                          GPUtype* waveVectors2,
                                          GPUtype cut2_chlk,
                                          GPUtype* position,
                                          GPUtype* charge,
                                          GPUtype* dcfLongRangeAsympK,
                                          GPUtype asympk_const,
                                          GPUtype smear2_4,
                                          int numAtoms,
                                          int numWaveVectors_2,
                                          int start_ind, 
                                          bool ionic, GPUtype* tcfLongRangeAsympK,
                                          GPUtype xappa2, GPUtype solvent_dielconst){

        int num_thread = 256;
        int num_blocks = (numWaveVectors_2 + num_thread - 1) / num_thread;

        // Launch the CUDA kernel
        K_calc_dcf_tcf_LongRangeAsympK<<<num_blocks, num_thread>>>(waveVectorX, waveVectorY, waveVectorZ, 
                                                     waveVectors2, cut2_chlk,
                                                     position, charge, 
                                                     dcfLongRangeAsympK,
                                                     asympk_const, smear2_4, 
                                                     numAtoms, numWaveVectors_2,
                                                     start_ind, ionic,
                                                     tcfLongRangeAsympK, xappa2,
                                                     solvent_dielconst);

        // Synchronize to ensure kernel completion
        hipDeviceSynchronize();

        // Serial version for debugging: I will keep it here for now
        // computeSumCosSin_serial(waveVectorX, waveVectorY, waveVectorZ, 
        //                         waveVectors2, cut2_chlk,
        //                         position, charge, 
        //                         dcfLongRangeAsympK, 
        //                         asympk_const, smear2_4,
        //                         numAtoms, numWaveVectors_2,
        //                         start_ind);

    }

    __global__ void K_calc_sum_cos_sin_huvk0_partial(GPUtype waveVectorX,
                                                    GPUtype waveVectorY,
                                                    GPUtype waveVectorZ,
                                                    GPUtype *positions,    
                                                    GPUtype *charges,      
                                                    int numAtoms,
                                                    GPUtype *blockCos,   // Partial sums (cosines)
                                                    GPUtype *blockSin) { // Partial sums (sines)
        extern __shared__ GPUPotAccumType sharedMemory[]; // Shared memory for partial sums
        GPUPotAccumType *sharedCos = sharedMemory;        // First half for cosines
        GPUPotAccumType *sharedSin = sharedMemory + blockDim.x; // Second half for sines

        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + tid;

        // Initialize shared memory
        sharedCos[tid] = 0.0;
        sharedSin[tid] = 0.0;

        // Each thread processes a subset of atoms
        for (int i = idx; i < numAtoms; i += blockDim.x * gridDim.x) {
            // GPUtype x = positions[3 * i];
            // GPUtype y = positions[3 * i + 1];
            // GPUtype z = positions[3 * i + 2];
            GPUtype x = positions[i];
            GPUtype y = positions[numAtoms + i];
            GPUtype z = positions[2 * numAtoms + i];
            GPUtype charge = charges[i];

            // Calculate phase
            GPUtype phase = waveVectorX * x + waveVectorY * y + waveVectorZ * z;

            // Accumulate cos and sin contributions
            sharedCos[tid] += static_cast<GPUPotAccumType>(charge * cos(phase));
            sharedSin[tid] -= static_cast<GPUPotAccumType>(charge * sin(phase));
        }

        __syncthreads();

        // Reduce partial sums within the block
        for (int s = blockDim.x / 2; s > 0; s /= 2) {
            if (tid < s) {
                sharedCos[tid] += sharedCos[tid + s];
                sharedSin[tid] += sharedSin[tid + s];
            }
            __syncthreads();
        }

        // Store the block's result in global memory
        if (tid == 0) {
            blockCos[blockIdx.x] = static_cast<GPUtype>(sharedCos[0]);
            blockSin[blockIdx.x] = static_cast<GPUtype>(sharedSin[0]);
        }
    }

    void rism3d_potential :: calc_sum_cos_sin_huvk0_serial(GPUtype waveVectorX_0,
                            GPUtype waveVectorY_0,
                            GPUtype waveVectorZ_0,
                            GPUtype* position,
                            GPUtype* charge,
                            int numAtoms,
                            GPUtype* sumcos_0,
                            GPUtype* sumsin_0){
        *sumcos_0 = 0.0;
        *sumsin_0 = 0.0;
        // int count = 0;
        for(int j = 0; j < numAtoms; j++){
            GPUtype phase = position[j]*waveVectorX_0 + position[j + numAtoms]*waveVectorY_0 + position[j + 2*numAtoms]*waveVectorZ_0;
            *sumcos_0 += charge[j] * cos(phase);
            *sumsin_0 += charge[j] * sin(phase);

            // // Printing a few values to check
            // if(count < 5){
            //     count += 1;
            //     cout << "==================================" << endl;
            //     cout << "index = " << j << endl;
            //     cout << "(x,y,z) = " << position[j] << " " << position[j + numAtoms] << " " << position[j + 2*numAtoms] << endl;
            //     cout << "(kx,ky,kz) = " << waveVectorX_0 << " " << waveVectorY_0 << " " << waveVectorZ_0 << endl;
            //     cout << "charge = " << charge[j] << endl;
            //     cout << "phase = " << phase << endl;
            //     cout << "charge[j] * cos(phase) = " << charge[j] * cos(phase) << endl;
            //     cout << "charge[j] * sin(phase) = " << charge[j] * sin(phase) << endl;
            //     cout << "==================================" << endl;
            // }
        }
    }

    void rism3d_potential :: calc_sum_cos_sin_huvk0(GPUtype waveVectorX_0,
                            GPUtype waveVectorY_0,
                            GPUtype waveVectorZ_0,
                            GPUtype* position,
                            GPUtype* charge,
                            int numAtoms,
                            GPUtype* sumcos_0,
                            GPUtype* sumsin_0){

        // Calling serial version to compare
        // calc_sum_cos_sin_huvk0_serial(waveVectorX_0, waveVectorY_0, waveVectorZ_0,
        //                               position, charge, numAtoms,
        //                               sumcos_0, sumsin_0);

        int numThreads = 256;
        int numBlocks = (numAtoms + numThreads - 1) / numThreads;

        int sharedMemSize = 2 * numThreads * sizeof(GPUPotAccumType);

        GPUtype *blockCos, *blockSin;
        hipMallocManaged((void**)&blockCos, numBlocks * sizeof(GPUtype));
        hipMallocManaged((void**)&blockSin, numBlocks * sizeof(GPUtype));

        *sumcos_0 = 0.0;
        *sumsin_0 = 0.0;
        K_calc_sum_cos_sin_huvk0_partial<<<numBlocks, numThreads, sharedMemSize>>>(waveVectorX_0, waveVectorY_0, waveVectorZ_0, 
                                                                                   position, charge, numAtoms, blockCos, blockSin);

        hipDeviceSynchronize();

        // Accumulate partial sums on the host
        for (int i = 0; i < numBlocks; ++i) {
            *sumcos_0 += blockCos[i];
            *sumsin_0 += blockSin[i];
        }

        // Free unified memory
        hipFree(blockCos);
        hipFree(blockSin);
    }

}