#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism3d.hpp"
using namespace std;

namespace rism3d_c{

    // Copy padded data into temporary array
    __global__ void getPadded(GPUtype *data, GPUtype *paddings, int Nx, int Ny, int Nz){
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < Nx * Ny * 2){
            int idx = gid / (Ny * 2);   // X index
            int idy = (gid / 2) % Ny;   // Y index
            int idz = gid % 2;          // Z index

            // Checking indexes
            // printf("idx (%d), idy (%d), idz (%d)\n", idx, idy, idz);

            int padding_idx = gid;
            int guv_padded_idx = idx * Ny * (Nz+2) + idy * (Nz+2) + Nz + idz;
            paddings[padding_idx] = data[guv_padded_idx];
        }

    }

    // Copy data trasnlating its positions into temporary array
    __global__ void translate_data_nr(GPUtype *data, GPUtype *data_temp, int Nx, int Ny, int Nz){
        int gid = blockIdx.x * blockDim.x + threadIdx.x; // x-axis

        if(gid < Nx * Ny * Nz){
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            // Checking indexes
            // printf("idx (%d), idy (%d), idz (%d)\n", idx, idy, idz);

            int fftw_id = idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz;

            int nr_id = gid;

            data_temp[nr_id] = data[fftw_id];
        }
    }

    __global__ void translate_data_fftw(GPUtype *data, GPUtype *data_temp, int Nx, int Ny, int Nz){
        int gid = blockIdx.x * blockDim.x + threadIdx.x; // x-axis

        if(gid < Nx * Ny * Nz){
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            // Checking indexes
            // printf("idx (%d), idy (%d), idz (%d)\n", idx, idy, idz);

            int fftw_id = idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz;

            int nr_id = gid;

            data_temp[fftw_id] = data[nr_id];
        }
    }

    __global__ void cpy_padded(GPUtype *data_temp, GPUtype *paddings, int Nx, int Ny, int Nz){
        int gid = blockIdx.x * blockDim.x + threadIdx.x; // x-axis

        if(gid < Nx * Ny * 2){
            int idx = gid / (Ny * 2);
            int idy = (gid / 2) % Ny;
            int idz = gid % 2;

            // Checking indexes
            // printf("idx (%d), idy (%d), idz (%d)\n", idx, idy, idz);

            int padding_idx = gid;
            int guv_padded_idx = idx * Ny * (Nz+2) + idy * (Nz+2) + Nz + idz;

            data_temp[guv_padded_idx] = paddings[padding_idx];
        }
    }

    __global__ void set_dcf_longrange(GPUtype* cuv, GPUtype solvent_charge, GPUtype* dcfLongRangeAsympR,
                                      int Nx, int Ny, int Nz){
        
        int igx = blockIdx.x * blockDim.x + threadIdx.x; 
        int igy = blockIdx.y * blockDim.y + threadIdx.y; 
        int igz = blockIdx.z * blockDim.z + threadIdx.z; 
        
        if (igx < Nx && igy < Ny && igz < Nz){
            int ig = igx * Ny * Nz + igy * Nz + igz;
            int padded_ig = igx * Ny * (Nz + 2) + igy * (Nz + 2) + igz;

            cuv[padded_ig] = solvent_charge * dcfLongRangeAsympR[ig];
        }

    }

    __global__ void subtract_dcf_longrange(GPUtype* guv,  GPUtype* cuv, 
                                           GPUtype solvent_charge, GPUtype* dcfLongRangeAsympR,
                                           int Nx, int Ny, int Nz){
        
        int igx = blockIdx.x * blockDim.x + threadIdx.x; 
        int igy = blockIdx.y * blockDim.y + threadIdx.y; 
        int igz = blockIdx.z * blockDim.z + threadIdx.z; 
        
        if (igx < Nx && igy < Ny && igz < Nz){
            int ig = igx * Ny * Nz + igy * Nz + igz;
            int padded_ig = igx * Ny * (Nz + 2) + igy * (Nz + 2) + igz;

            guv[padded_ig] = cuv[padded_ig] - solvent_charge * dcfLongRangeAsympR[ig];
        }

    }

    __global__ void add_dcf_longrange(GPUtype* guv, GPUtype solvent_charge, 
                                      GPUtype* dcfLongRangeAsympK,
                                      int N){
        
        int ig = blockIdx.x * blockDim.x + threadIdx.x; 
        
        if (ig < N){
            guv[ig] = guv[ig] - solvent_charge * dcfLongRangeAsympK[ig];
        }

    }

    __global__ void subtract_tcf_longrange(GPUtype* huv, GPUtype solvent_charge_sp, 
                                           GPUtype* tcfLongRangeAsympK,
                                           int N){
        
        // As in the fortran code, we are skipping the real and imaginary part for
        // k = 0
        int ig = blockIdx.x * blockDim.x + threadIdx.x + 2; 
        
        if (ig < N){
            huv[ig] = huv[ig] + solvent_charge_sp * tcfLongRangeAsympK[ig];
        }

    }

    void rism3d :: set_dcf_longrange_cu(GPUtype* cuv, GPUtype* solvent_charge, GPUtype* dcfLongRangeAsympR){
        int Nx = grid.globalDimsR[0];
        int Ny = grid.globalDimsR[1];
        int Nz = grid.globalDimsR[2];

        dim3 blockDim(8, 8, 8);
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y,
            (Nz + blockDim.z - 1) / blockDim.z
        );

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            set_dcf_longrange<<<gridDim, blockDim>>>(cuv + iv * Nx * Ny * (Nz + 2), 
                                                     solvent_charge[iv], dcfLongRangeAsympR,
                                                     Nx, Ny, Nz);
        }
        hipDeviceSynchronize();

    }

    void rism3d :: subtract_dcf_longrange_cu(GPUtype* guv, GPUtype* cuv, GPUtype* solvent_charge, GPUtype* dcfLongRangeAsympR){
        int Nx = grid.globalDimsR[0];
        int Ny = grid.globalDimsR[1];
        int Nz = grid.globalDimsR[2];

        dim3 blockDim(8, 8, 8);
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y,
            (Nz + blockDim.z - 1) / blockDim.z
        );

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            subtract_dcf_longrange<<<gridDim, blockDim>>>(guv + iv * Nx * Ny * (Nz + 2),
                                                          cuv + iv * Nx * Ny * (Nz + 2), 
                                                          solvent_charge[iv], dcfLongRangeAsympR,
                                                          Nx, Ny, Nz);
        }
        hipDeviceSynchronize();

    }

    __global__ void add_tcf_longrange(GPUtype* huv,  GPUtype solvent_charge_sp, 
                                      GPUtype* tcfLongRangeAsympR,
                                      int Nx, int Ny, int Nz){
        
        int igx = blockIdx.x * blockDim.x + threadIdx.x; 
        int igy = blockIdx.y * blockDim.y + threadIdx.y; 
        int igz = blockIdx.z * blockDim.z + threadIdx.z; 
        
        if (igx < Nx && igy < Ny && igz < Nz){
            int ig = igx * Ny * Nz + igy * Nz + igz;
            int padded_ig = igx * Ny * (Nz + 2) + igy * (Nz + 2) + igz;

            huv[padded_ig] = huv[padded_ig] + solvent_charge_sp * tcfLongRangeAsympR[ig];
        }

    }

    void rism3d :: add_dcf_longrange_cu(GPUtype* guv, GPUtype* solvent_charge, GPUtype* dcfLongRangeAsympK){
        int totalLocalPointsK = grid.totalLocalPointsK;

        int num_threads = 256;
        int num_blocks = (totalLocalPointsK + num_threads-1) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            add_dcf_longrange<<<num_blocks, num_threads>>>(guv + iv * totalLocalPointsK,
                                                           solvent_charge[iv], dcfLongRangeAsympK,
                                                           totalLocalPointsK);
        }
        hipDeviceSynchronize();

    }

    void rism3d :: subtract_tcf_longrange_cu(GPUtype* huv, GPUtype* solvent_charge_sp, GPUtype* tcfLongRangeAsympK){
        int totalLocalPointsK = grid.totalLocalPointsK;

        int num_threads = 256;
        int num_blocks = (totalLocalPointsK + num_threads-1) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            subtract_tcf_longrange<<<num_blocks, num_threads>>>(huv + iv * totalLocalPointsK,
                                                                solvent_charge_sp[iv], tcfLongRangeAsympK,
                                                                totalLocalPointsK);
        }
        hipDeviceSynchronize();

    }

    void rism3d :: add_tcf_longrange_cu(GPUtype* huv, GPUtype* solvent_charge_sp, GPUtype* tcfLongRangeAsympR){
        int Nx = grid.globalDimsR[0];
        int Ny = grid.globalDimsR[1];
        int Nz = grid.globalDimsR[2];

        dim3 blockDim(8, 8, 8);
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y,
            (Nz + blockDim.z - 1) / blockDim.z
        );

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            add_tcf_longrange<<<gridDim, blockDim>>>(huv + iv * Nx * Ny * (Nz + 2),
                                                     solvent_charge_sp[iv], tcfLongRangeAsympR,
                                                     Nx, Ny, Nz);
        }
        hipDeviceSynchronize();

    }

    void rism3d :: convert2nr_cu(){
        // create array to store non-padded data
        array_class<GPUtype> guv_temp(array_class<GPUtype>::ROW_MAJOR);
        guv_temp.set_memalloc(&memalloc, true);
        guv_temp.alloc_mem(solventclass.numAtomTypes, grid.globalDimsR[0], grid.globalDimsR[1], grid.globalDimsR[2]);

        // create array store extra padding results
        array_class<GPUtype> paddings(array_class<GPUtype>::ROW_MAJOR);
        paddings.set_memalloc(&memalloc, true);
        paddings.alloc_mem(solventclass.numAtomTypes, grid.globalDimsR[0], grid.globalDimsR[1], 2);

        // Dimensions
        int Nx = grid.globalDimsR[0];
        int Ny = grid.globalDimsR[1];
        int Nz = grid.globalDimsR[2];
        int Nz_pad = Nz + 2;

        // Define the number of threads and blocks to be used
        int num_threads = 256;
        int num_blocks = ((Nx * Ny * 2) + (num_threads-1)) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            getPadded<<<num_blocks, num_threads>>>(guv.m_data + iv * Nx * Ny * Nz_pad, 
                                                   paddings.m_data + iv * Nx * Ny * 2, 
                                                   Nx, Ny, Nz);   
            
            // Checking errors while launching kernel
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                cout << "getPadded kernel launch failed: " << hipGetErrorString(err) << endl;
                abort();
            }
        }

        // Redefine number of blocks to be used in next step
        num_blocks = ((Nx * Ny * Nz) + (num_threads-1)) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            translate_data_nr<<<num_blocks, num_threads>>>(guv.m_data + iv * Nx * Ny * Nz_pad, 
                                                           guv_temp.m_data + iv * Nx * Ny * Nz, 
                                                           Nx, Ny, Nz);

            // Checking errors while launching kernel
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                cout << "translate_data_nr kernel launch failed: " << hipGetErrorString(err) << endl;
                abort();
            }
        }

        // Ensure all data is in temporary arrays before copying them back to data array
        hipDeviceSynchronize();

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            // Here we can use hipMemcpyAsync to copy everything simultaneously
            hipMemcpyAsync(guv.m_data + iv * Nx * Ny * (Nz + 2), guv_temp.m_data + iv * Nx * Ny * Nz, Nx * Ny * Nz * sizeof(GPUtype), hipMemcpyDeviceToDevice);
            hipMemcpyAsync(guv.m_data + iv * Nx * Ny * (Nz + 2) + Nx * Ny * Nz, paddings.m_data + iv * 2 * Nx * Ny, 2 * Nx * Ny * sizeof(GPUtype), hipMemcpyDeviceToDevice);
        }

        // Ensure all data was copied back to data array before proceeding with calculations
        hipDeviceSynchronize();
        
    }

    void rism3d :: convert2fftw_cu(){
        // create array to store non-padded data
        array_class<GPUtype> huv_temp(array_class<GPUtype>::ROW_MAJOR);
        huv_temp.set_memalloc(&memalloc, true);
        huv_temp.alloc_mem(solventclass.numAtomTypes, grid.globalDimsR[0], grid.globalDimsR[1], grid.globalDimsR[2] + 2);

        // create array store extra padding results
        array_class<GPUtype> paddings(array_class<GPUtype>::ROW_MAJOR);
        paddings.set_memalloc(&memalloc, true);
        paddings.alloc_mem(solventclass.numAtomTypes, grid.globalDimsR[0], grid.globalDimsR[1], 2);

        // Dimensions
        int Nx = grid.globalDimsR[0];
        int Ny = grid.globalDimsR[1];
        int Nz = grid.globalDimsR[2];
        // int Nz_pad = Nz + 2;

        // Define the number of threads and blocks to be used
        int num_threads = 256;
        int num_blocks = ((Nx * Ny * Nz) + (num_threads-1)) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            // copying padded data into paddings
            hipMemcpyAsync(paddings.m_data + iv * 2 * Nx * Ny, huv.m_data + iv * Nx * Ny * (Nz + 2) + Nx * Ny * Nz, 2 * Nx * Ny * sizeof(GPUtype), hipMemcpyDeviceToDevice);

            // translating data to be in fftw format
            translate_data_fftw<<<num_blocks,num_threads>>>(huv.m_data + iv * Nx * Ny * (Nz + 2), 
                                                            huv_temp.m_data + iv * Nx * Ny * (Nz + 2), 
                                                            Nx, Ny, Nz);
            
            // checking for errors while lauching kernel
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                cout << "translate_data_fftw kernel launch failed: " << hipGetErrorString(err) << endl;
                abort();
            }
        }

        // Ensure data is translated and padded data is copied into temporary array before proceeding
        hipDeviceSynchronize();

        // copying data from temp array into actual data array
        hipMemcpy(huv.m_data, huv_temp.m_data, solventclass.numAtomTypes * Nx * Ny * (Nz+2) * sizeof(GPUtype), hipMemcpyDeviceToDevice);
        
        // redefine number of blocks to be used
        num_blocks = ((Nx * Ny * 2) + (num_threads-1)) / num_threads;

        // copying padded data back into data array
        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            cpy_padded<<<num_blocks, num_threads>>>(huv.m_data + iv * Nx * Ny * (Nz + 2), 
                                                    paddings.m_data + iv * Nx * Ny * 2, 
                                                    Nx, Ny, Nz);   

            // checking for errors while lauching kernel
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                cout << "cpy_padded kernel launch failed: " << hipGetErrorString(err) << endl;
                abort();
            }
        }

        // Ensure all data is back into data array before proceeding with calculations
        hipDeviceSynchronize();

    }

    __global__ void set_padding2zero_cu(GPUtype *data, int Nx, int Ny, int Nz){
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < Nx * Ny * 2){
            int idx = gid / (Ny * 2);   // X index
            int idy = (gid / 2) % Ny;   // Y index
            int idz = gid % 2;          // Z index

            int guv_padded_idx = idx * Ny * (Nz+2) + idy * (Nz+2) + Nz + idz;
            data[guv_padded_idx] = 0;
        }

    }

    void rism3d :: set_padding2zero(GPUtype* data){
        int num_threads = 256;
        int num_blocks = ((grid.globalDimsR[0] * grid.globalDimsR[1] * 2) + (num_threads-1)) / num_threads;

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            set_padding2zero_cu<<<num_blocks, num_threads>>>(data + iv * grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2), 
                                                   grid.globalDimsR[0], grid.globalDimsR[1], grid.globalDimsR[2]);   
            
            // Checking errors while launching kernel
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                cout << "getPadded kernel launch failed: " << hipGetErrorString(err) << endl;
                abort();
            }
        }

        // Ensure all padded values are set to zero before proceeding
        hipDeviceSynchronize();

    }

    __global__ void get_residue_cu(GPUtype *residue_array, GPUtype *new_data, GPUtype *old_data, 
                                   int nat, int Nx, int Ny, int Nz, int Nz_pad){
        int gid = threadIdx.x + blockDim.x * blockIdx.x;

        if(gid < nat * Nx * Ny * Nz){
            int iv = gid / (Nx * Ny * Nz);
            int idx = (gid / (Ny * Nz)) % Nx;
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            int id = iv * Nx * Ny * Nz_pad + idx * Ny * Nz_pad + idy * Nz_pad + idz;

            residue_array[id] = (new_data[id] - 1) - old_data[id];

        }

    }

    void rism3d :: get_residue(){
        int num_threads = 256;
        int num_blocks = ((solventclass.numAtomTypes * grid.globalDimsR[0] * grid.globalDimsR[1] * grid.globalDimsR[2]) + (num_threads-1)) / num_threads;

        get_residue_cu<<<num_blocks, num_threads>>>(cuvres.m_data, guv.m_data, huv.m_data,
                                                    solventclass.numAtomTypes, grid.globalDimsR[0],
                                                    grid.globalDimsR[1], grid.globalDimsR[2],
                                                    grid.globalDimsR[2] + 2);
        hipDeviceSynchronize();

    }

    __global__ void get_h_k_cu(GPUtype *huv, GPUtype *guv, GPUtype *xvva, int *waveVectorWaveNumberMap, 
                               int totalLocalPointsK){
        int gid = threadIdx.x + blockIdx.x * blockDim.x;

        if(gid < totalLocalPointsK){
            int iga = waveVectorWaveNumberMap[gid/2];

            huv[gid] = huv[gid] + guv[gid] * xvva[iga];
        }

    }

    void rism3d :: get_h_k(){
        hipMemset(huv.m_data, 0, solventclass.numAtomTypes*grid.totalLocalPointsK*sizeof(GPUtype));
        
        int num_threads = 256;
        int num_blocks = (grid.totalLocalPointsK + (num_threads-1)) / num_threads;

        for(int iv2 = 0; iv2 < solventclass.numAtomTypes; iv2++){
            for(int iv1 = 0; iv1 < solventclass.numAtomTypes; iv1++){
                get_h_k_cu<<<num_blocks, num_threads>>>(huv.m_data + iv1 * grid.totalLocalPointsK,
                                                        guv.m_data + iv2 * grid.totalLocalPointsK,
                                                        xvva.m_data + iv1 * solventclass.numAtomTypes * grid.waveNumberArraySize + iv2 * grid.waveNumberArraySize,
                                                        grid.waveVectorWaveNumberMap.m_data,
                                                        grid.totalLocalPointsK
                                                        );
            }
            hipDeviceSynchronize();
        }
        // Do we need this second sybchronization or the one before would handle all synchronization?
        // I guess it would handle every synchronization needed
        // hipDeviceSynchronize();    
    }

    ///////// Conversion between numerical recipes and fftw layout: CPU version

    void rism3d :: convert2nr(){
        ////////// CHANGING TO NR SCHEME
        // create array to point extra padding results
        array_class<GPUtype> paddings;
        paddings.set_memalloc(&memalloc);
        paddings.alloc_mem(grid.globalDimsR[0], grid.globalDimsR[1], 2);

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            // copy padded values into paddings array
            for(int ix = 0; ix < grid.globalDimsR[0]; ix++){
                for(int iy = 0; iy < grid.globalDimsR[1]; iy++){
                    for(int iz = 0; iz < 2; iz++){
                        paddings.m_data[ix*grid.globalDimsR[1]*2 + iy*2 + iz] = guv.m_data[iv*grid.globalDimsR[0]*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) + 
                                                                        ix*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) +
                                                                        iy*(grid.globalDimsR[2] + 2) + grid.globalDimsR[2] + iz];
                    }
                }
            }

            // move non-padded data to be stored contiguously
            for(int ix = 0; ix < grid.globalDimsR[0]; ix++){
                for(int iy = 0; iy < grid.globalDimsR[1]; iy++){
                    
                    int fftw_id = iv*grid.globalDimsR[0]*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) + 
                                    ix*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) +
                                    iy*(grid.globalDimsR[2] + 2);

                    // The numerical recipes layout is:
                    // "a contiguous piece of nx*ny*nz memory, with the Nyquist
                    // frequency data (2*ny*nz) starting at the end of this block"
                    // obs: this is true for each atom type. Thus, the numerical recipe index (nr_id)
                    // should still stride considering the paddings when looping over the atom type 
                    // index (iv) to make sure the xyz block starts at the correct positions for
                    // each atom type
                    int nr_id = iv*grid.globalDimsR[0]*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) + 
                                ix*grid.globalDimsR[1]*grid.globalDimsR[2] +
                                iy*grid.globalDimsR[2];
                    
                    for(int iz = 0; iz < grid.globalDimsR[2]; iz++){
                        guv.m_data[nr_id + iz] = guv.m_data[fftw_id + iz];
                    }
                }
            }
        
            // Calculate index where padded values should start
            int end_id = iv*(grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2)) + 
                            grid.globalDimsR[0]*grid.globalDimsR[1]*grid.globalDimsR[2];
            
            // Add padded values at the end of each chunk of memory containing data for each solvent index
            for(int i = 0; i < grid.globalDimsR[0]*grid.globalDimsR[1]*2; i++){
                // cout << "end_id = " << end_id << endl;
                guv.m_data[end_id + i] = paddings.m_data[i];
            }
        }
    }

    void rism3d :: convert2fftw(){

        ////////// CHANGING TO NR SCHEME
        // create array to point extra padding results
        array_class<GPUtype> paddings;
        paddings.set_memalloc(&memalloc);
        paddings.alloc_mem(grid.globalDimsR[0], grid.globalDimsR[1], 2);

        array_class<GPUtype> huv_new;
        huv_new.set_memalloc(&memalloc);
        huv_new.alloc_mem(solventclass.numAtomTypes, grid.globalDimsR[0], grid.globalDimsR[1], grid.globalDimsR[2] + 2);

        for(int iv = 0; iv < solventclass.numAtomTypes; iv++){
            for(int ix = 0; ix < grid.globalDimsR[0]; ix++){
                for(int iy = 0; iy < grid.globalDimsR[1]; iy++){
                    for (int iz = 0; iz < grid.globalDimsR[2]; iz++) {
                        huv_new.m_data[iv * grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2) + 
                                       ix * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2) + 
                                       iy * (grid.globalDimsR[2] + 2) + iz] = 
                                       huv.m_data[iv * grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2) + 
                                                  ix * grid.globalDimsR[1] * grid.globalDimsR[2] + iy * grid.globalDimsR[2] + iz];
                    }
                }
            }

            // Calculate index where padded values starts
            int end_id = iv*(grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2)) + 
                            grid.globalDimsR[0]*grid.globalDimsR[1]*grid.globalDimsR[2];
            
            // Copy padded values into padding array
            for(int i = 0; i < grid.globalDimsR[0]*grid.globalDimsR[1]*2; i++){
                paddings.m_data[i] = huv.m_data[end_id + i];
            }
            
            // Place padded values at the correct places
            for(int ix = 0; ix < grid.globalDimsR[0]; ix++){
                for(int iy = 0; iy < grid.globalDimsR[1]; iy++){
                    for(int iz = 0; iz < 2; iz++){
                        huv_new.m_data[iv*grid.globalDimsR[0]*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) + 
                                       ix*grid.globalDimsR[1]*(grid.globalDimsR[2] + 2) +
                                       iy*(grid.globalDimsR[2] + 2) + grid.globalDimsR[2] + iz] = 
                                       paddings.m_data[ix*grid.globalDimsR[1]*2 + iy*2 + iz];
                    }
                }
            }
        }

        int size = solventclass.numAtomTypes * grid.globalDimsR[0] * grid.globalDimsR[1] * (grid.globalDimsR[2] + 2);
        copy(huv_new.m_data, huv_new.m_data + size, huv.m_data);
    }



}