#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism_util.hpp"
using namespace std;

#if defined(__CUDA_ARCH__ ) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

namespace rism3d_c {

    __device__ void polynomialInterpolation(GPUtype *x0, GPUtype* y0, int n, GPUtype x, GPUtype *y){
        GPUtype *p = new GPUtype[n];
        // Copy values from y0 into p
        for(int j = 0; j < n; j++){
            p[j] = y0[j];
        }
        
        for(int m = 0; m < n - 2; m++){
            for(int i = 0; i < n-m-1; i++){
                p[i] = (x - x0[i + m + 1]) * p[i] - (x - x0[i]) * p[i+1];
                p[i] = p[i] / (x0[i] - x0[i+m + 1]);
            }
        }
        
        *y = ((x - x0[n-1]) * p[0] - (x - x0[0]) * p[1]) / (x0[0] - x0[n-1]);

        GPUtype error = (p[0] + p[1] - 2.0f * (*y)) / 2.0f;
        if(error > 1e-4){
            printf("Error = %f. Greater than 1e-4! \n", error);
        }
        delete p;
    
    }

    __global__ void polynomialInterpolation_kernel(int maxPointsToInterp, int numPoints, int numPointsToInterp,
                                            GPUtype *x0, GPUtype *y0, GPUtype *x, GPUtype *y, int iv1, int iv2, int Nyz){
        // numPoints == numRDFpoints
        // numPointsToInterp == waveNumberArraySize
        // x0 == solventWaveNumbers
        // y0 == xvv
        // x == gridWaveNumbers
        // y == xvva
        // Nyz = number of solvent atom types
        

        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        if(idx < numPointsToInterp){
            int igk1 = 1;

            for(int igk = 0; igk < numPoints - maxPointsToInterp + 1; igk++){
                igk1 = igk;
                if(x0[igk1 + maxPointsToInterp/2] > x[idx]){
                    break;
                }
            }

            polynomialInterpolation(x0 + igk1, y0 + igk1, maxPointsToInterp, x[idx], &y[iv1*Nyz*numPointsToInterp + iv2*numPointsToInterp + idx]);

        }

    }

    // Templates for cu_memcpy. These allow us copy memory to variables with different precision

    template <typename SrcType, typename DstType>
    __global__ void memcpy_kernel(const SrcType *src, DstType *dst, int size){
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(idx < size){
            dst[idx] = src[idx];
        }
    }

    template __global__ void memcpy_kernel<float, float>(const float *src, float *dst, int size);
    template __global__ void memcpy_kernel<double, double>(const double *src, double *dst, int size);
    template __global__ void memcpy_kernel<double, float>(const double *src, float *dst, int size);
    template __global__ void memcpy_kernel<float, double>(const float *src, double *dst, int size);

    template <typename SrcType, typename DstType>
    void cu_memcpy(const SrcType *src, DstType *dst, int size){
        int num_blocks = (size + 255) / 256;
        int num_threads = 256;

        memcpy_kernel<SrcType,DstType><<<num_blocks, num_threads>>>(src, dst, size);
        hipDeviceSynchronize();
    }

    void cu_memcpy(const float *src, float *dst, int size) {
        cu_memcpy<float, float>(src, dst, size);
    }
    
    void cu_memcpy(const double *src, double *dst, int size) {
        cu_memcpy<double, double>(src, dst, size);
    }
    
    void cu_memcpy(const double *src, float *dst, int size) {
        cu_memcpy<double, float>(src, dst, size);
    }
    
    void cu_memcpy(const float *src, double *dst, int size) {
        cu_memcpy<float, double>(src, dst, size);
    }
    
    __global__ void reduction_kernel(GPUtype* input, GPUReduceAccumType* output, int N) {
        extern __shared__ GPUReduceAccumType shared_data[];

        int idx = threadIdx.x + blockIdx.x * blockDim.x; // global index
        int tid = threadIdx.x; // thread's local id within the block (ranging from 0 to blockDim.x - 1).

        // Load elements into shared memory
        if (idx < N) {
            shared_data[tid] = static_cast<GPUReduceAccumType>(input[idx]);
        } else {
            shared_data[tid] = 0.0;//f;
        }

        __syncthreads();  // Synchronize threads in the block

        // Perform parallel reduction in shared memory:
        // This step will sum pair of values while reduicing the
        // the number of elements to be computed. Ex.: for the first block
        // containing 8 values
        // arr = [0 1 2 3 4 5 6 7]
        // for stride = 1:
        //     tid = 0, 2, 4, 6
        //     arr = [1 1 5 3 9 5 13 7]
        // for stride = 2:
        //     tid = 0, 4
        //     arr = [6 1 5 3 22 5 13 7]
        // for stride = 4
        //     tid = 0
        //     arr = [28 1 5 3 22 5 13 7]
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            if (tid % (2 * stride) == 0) {
                shared_data[tid] += shared_data[tid + stride];
            }
            __syncthreads();  // Ensure all threads have updated shared memory
        }

        // Write the result of the reduction in the first thread of the block
        // The value of the first thread of each block (which stores the total
        // sum for that block) will be asigned to the output array at the block 
        // id position
        if (tid == 0) {
            // output[blockIdx.x] = static_cast<GPUtype>(shared_data[0]);
            output[blockIdx.x] = shared_data[0];
        }
    }

    void test(GPUtype *xvv){
        for(int i = 0; i < 100; i++){
            cout << xvv[i] << endl;
        }
    }

    void cu_polinomialInterpolation(int maxPointsToInterp, int waveNumberArraySize, 
                                    int numRDFpoints, int numAtomTypes,
                                    GPUtype *solventWaveNumbers, 
                                    GPUtype *xvv, GPUtype *gridWaveNumbers,
                                    GPUtype *xvva, int iv1, int iv2, int Nyz){
        

        ////////////////////////////////////////////////////////////////////////////////////////
        int num_blocks = (waveNumberArraySize+255)/256;
        int num_threads = 256;

        // test(xvv);

        // int device = -1;
        // hipGetDevice(&device);

        // hipMemPrefetchAsync(solventWaveNumbers, numRDFpoints*sizeof(GPUtype),device,NULL);
        // hipMemPrefetchAsync(xvv, numRDFpoints*numAtomTypes*numAtomTypes*sizeof(GPUtype),device,NULL);
        // hipMemPrefetchAsync(gridWaveNumbers, waveNumberArraySize*sizeof(GPUtype),device,NULL);

        // // Here I think we can just reserve the ammount of memory for each pair of solvent atom types,
        // // since we do not need memory beyond that for each kernel call and each kernel call will start at
        // // the correct contigous memory address.
        // hipMemPrefetchAsync(xvva, waveNumberArraySize*sizeof(GPUtype),device,NULL);

        polynomialInterpolation_kernel<<<num_blocks,num_threads>>>(maxPointsToInterp, numRDFpoints, 
                                                                   waveNumberArraySize, 
                                                                   solventWaveNumbers, 
                                                                   xvv, 
                                                                   gridWaveNumbers, 
                                                                   xvva, iv1, iv2, Nyz);
        
    }

    GPUtype cu_reduce_sum(GPUtype* arr, int N){
        int block_size = 256; // number of threads in each block
        int grid_size = (N + block_size - 1) / block_size; // total number of blocks to be used
        int shared_memory_size = block_size * sizeof(GPUtype);

        GPUReduceAccumType *output;
        hipMallocManaged(&output, (N + 255) / 256 * sizeof(GPUtype));  // One per block
        hipMemset(output, 0, (N + 255) / 256 * sizeof(GPUtype));

        // Launch the kernel
        // This will ensure that there will be lauched grid_size blocks,
        // each containing 256 threads and using enough shared memory to 
        // store shared_memory_size float values
        reduction_kernel<<<grid_size, block_size, shared_memory_size>>>(arr, output, N);

        hipDeviceSynchronize();  // Wait for the kernel to finish

        // Final reduction on the host
        GPUReduceAccumType final_sum = 0.0;
        for (int i = 0; i < grid_size; i++) {
            final_sum += output[i];
        }

        // Cleanup
        hipFree(output);

        return static_cast<GPUtype>(final_sum);
    }

    // Helper inline functions for FMA in device code.
    __device__ inline float myFMA(float a, float b, float c) {
        return __fmaf_rn(a, b, c);  // FMA in single precision
    }

    __device__ inline double myFMA(double a, double b, double c) {
        return __fma_rn(a, b, c);   // FMA in double precision
    }


    // Kernel to compute dot product with mixed precision.
    // Input arrays x and y are in single precision, but the accumulation is either float or double.
    // It may be possible to use a template this but the shared is problematic.
    __global__ void SDDotKernel(const float* x, const float* y, int n, float* result) {
        extern __shared__ float share_floats[];
        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + tid;
        float sum = 0.0;
        
        // Each thread processes multiple elements if necessary.
        while (idx < n) {
            // sum += (float)x[idx] * (float)y[idx];
            sum = myFMA(x[idx], y[idx], sum);
            idx += blockDim.x * gridDim.x;
        }
        share_floats[tid] = sum;
        __syncthreads();
        
        // Reduction in shared memory.
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                share_floats[tid] += share_floats[tid + s];
            }
            __syncthreads();
        }
        
        // The first thread in each block atomically adds the block's sum to the global result.
        if (tid == 0) {
            atomicAdd(result, share_floats[0]);
        }
    }

    __global__ void SDDotKernel(const float* x, const float* y, int n, double* result) {
        extern __shared__ double share_doubles[];
        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + tid;
        double sum = 0.0;
        
        // Each thread processes multiple elements if necessary.
        while (idx < n) {
            // sum += (double)x[idx] * (double)y[idx];
            sum = myFMA((double)x[idx], (double)y[idx], sum);
            idx += blockDim.x * gridDim.x;
        }
        share_doubles[tid] = sum;
        __syncthreads();
        
        // Reduction in shared memory.
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                share_doubles[tid] += share_doubles[tid + s];
            }
            __syncthreads();
        }
        
        // The first thread in each block atomically adds the block's sum to the global result.
        if (tid == 0) {
            atomicAdd(result, share_doubles[0]);
        }
    }

    // Mixed precision dot product: single precision inputs, double precision result.
    template <typename SrcType, typename DstType>
    void cu_SDDot(int n, const SrcType* d_x, const SrcType* d_y, DstType* d_result) {
        int blockSize = 256;
        int gridSize = (n + blockSize - 1) / blockSize;
        
        // Initialize the result to 0.
        hipMemset(d_result, 0, sizeof(DstType));
        
        // Launch the kernel. Allocate shared memory for block reduction.
        SDDotKernel<<<gridSize, blockSize, blockSize * sizeof(DstType)>>>(d_x, d_y, n, d_result);
        
        // Synchronize to ensure the kernel has completed.
        hipDeviceSynchronize();
    }
    void cu_SDDot(int n, const float* d_x, const float* d_y, float* d_result) {
        cu_SDDot<float, float>(n, d_x, d_y, d_result);
    }
    void cu_SDDot(int n, const float* d_x, const float* d_y, double* d_result) {
        cu_SDDot<float, double>(n, d_x, d_y, d_result);
    }
    // void cu_SDDot(int n, const double* d_x, const double* d_y, double* d_result) {
    //     cu_SDDot<double, double>(n, d_x, d_y, d_result);
    // }
}