#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism_util.hpp"
using namespace std;

namespace rism3d_c {

    __device__ void polynomialInterpolation(GPUtype *x0, GPUtype* y0, int n, GPUtype x, GPUtype *y){
        GPUtype *p = new GPUtype[n];
        // Copy values from y0 into p
        for(int j = 0; j < n; j++){
            p[j] = y0[j];
        }
        
        for(int m = 0; m < n - 2; m++){
            for(int i = 0; i < n-m-1; i++){
                p[i] = (x - x0[i + m + 1]) * p[i] - (x - x0[i]) * p[i+1];
                p[i] = p[i] / (x0[i] - x0[i+m + 1]);
            }
        }
        
        *y = ((x - x0[n-1]) * p[0] - (x - x0[0]) * p[1]) / (x0[0] - x0[n-1]);

        GPUtype error = (p[0] + p[1] - 2.0f * (*y)) / 2.0f;
        if(error > 1e-4){
            printf("Error = %f. Greater than 1e-4! \n", error);
        }
        delete p;
    
    }

    __global__ void polynomialInterpolation_kernel(int maxPointsToInterp, int numPoints, int numPointsToInterp,
                                            GPUtype *x0, GPUtype *y0, GPUtype *x, GPUtype *y, int iv1, int iv2, int Nyz){
        // numPoints == numRDFpoints
        // numPointsToInterp == waveNumberArraySize
        // x0 == solventWaveNumbers
        // y0 == xvv
        // x == gridWaveNumbers
        // y == xvva
        // Nyz = number of solvent atom types
        

        int idx = threadIdx.x + blockIdx.x * blockDim.x;

        if(idx < numPointsToInterp){
            int igk1 = 1;

            for(int igk = 0; igk < numPoints - maxPointsToInterp + 1; igk++){
                igk1 = igk;
                if(x0[igk1 + maxPointsToInterp/2] > x[idx]){
                    break;
                }
            }

            polynomialInterpolation(x0 + igk1, y0 + igk1, maxPointsToInterp, x[idx], &y[iv1*Nyz*numPointsToInterp + iv2*numPointsToInterp + idx]);

        }

    }

    __global__ void memcpy_kernel(GPUtype *src, GPUtype *dst, int size){
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        
        if(idx < size){
            dst[idx] = src[idx];
        }
    }

    __global__ void reduction_kernel(GPUtype* input, GPUtype* output, int N) {
        extern __shared__ GPUtype shared_data[];

        int idx = threadIdx.x + blockIdx.x * blockDim.x; // global index
        int tid = threadIdx.x; // thread's local id within the block (ranging from 0 to blockDim.x - 1).

        // Load elements into shared memory
        if (idx < N) {
            shared_data[tid] = input[idx];
        } else {
            shared_data[tid] = 0.0f;
        }

        __syncthreads();  // Synchronize threads in the block

        // Perform parallel reduction in shared memory:
        // This step will sum pair of values while reduicing the
        // the number of elements to be computed. Ex.: for the first block
        // containing 8 values
        // arr = [0 1 2 3 4 5 6 7]
        // for stride = 1:
        //     tid = 0, 2, 4, 6
        //     arr = [1 1 5 3 9 5 13 7]
        // for stride = 2:
        //     tid = 0, 4
        //     arr = [6 1 5 3 22 5 13 7]
        // for stride = 4
        //     tid = 0
        //     arr = [28 1 5 3 22 5 13 7]
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            if (tid % (2 * stride) == 0) {
                shared_data[tid] += shared_data[tid + stride];
            }
            __syncthreads();  // Ensure all threads have updated shared memory
        }

        // Write the result of the reduction in the first thread of the block
        // The value of the first thread of each block (which stores the total
        // sum for that block) will be asigned to the output array at the block 
        // id position
        if (tid == 0) {
            output[blockIdx.x] = shared_data[0];
        }
    }

    void test(GPUtype *xvv){
        for(int i = 0; i < 100; i++){
            cout << xvv[i] << endl;
        }
    }

    void cu_memcpy(GPUtype *src, GPUtype *dst, int size){
        int num_blocks = (size + 255) / 256;
        int num_threads = 256;

        memcpy_kernel<<<num_blocks, num_threads>>>(src, dst, size);
        hipDeviceSynchronize();
    }

    void cu_polinomialInterpolation(int maxPointsToInterp, int waveNumberArraySize, 
                                    int numRDFpoints, int numAtomTypes,
                                    GPUtype *solventWaveNumbers, 
                                    GPUtype *xvv, GPUtype *gridWaveNumbers,
                                    GPUtype *xvva, int iv1, int iv2, int Nyz){
        

        ////////////////////////////////////////////////////////////////////////////////////////
        int num_blocks = (waveNumberArraySize+255)/256;
        int num_threads = 256;

        // test(xvv);

        // int device = -1;
        // hipGetDevice(&device);

        // hipMemPrefetchAsync(solventWaveNumbers, numRDFpoints*sizeof(GPUtype),device,NULL);
        // hipMemPrefetchAsync(xvv, numRDFpoints*numAtomTypes*numAtomTypes*sizeof(GPUtype),device,NULL);
        // hipMemPrefetchAsync(gridWaveNumbers, waveNumberArraySize*sizeof(GPUtype),device,NULL);

        // // Here I think we can just reserve the ammount of memory for each pair of solvent atom types,
        // // since we do not need memory beyond that for each kernel call and each kernel call will start at
        // // the correct contigous memory address.
        // hipMemPrefetchAsync(xvva, waveNumberArraySize*sizeof(GPUtype),device,NULL);

        polynomialInterpolation_kernel<<<num_blocks,num_threads>>>(maxPointsToInterp, numRDFpoints, 
                                                                   waveNumberArraySize, 
                                                                   solventWaveNumbers, 
                                                                   xvv, 
                                                                   gridWaveNumbers, 
                                                                   xvva, iv1, iv2, Nyz);
        
    }

    GPUtype cu_reduce_sum(GPUtype* arr, int N){
        int block_size = 256; // number of threads in each block
        int grid_size = (N + block_size - 1) / block_size; // total number of blocks to be used
        int shared_memory_size = block_size * sizeof(GPUtype);

        GPUtype *output;
        hipMallocManaged(&output, (N + 255) / 256 * sizeof(GPUtype));  // One per block

        // Launch the kernel
        // This will ensure that there will be lauched grid_size blocks,
        // each containing 256 threads and using enough shared memory to 
        // store shared_memory_size float values
        reduction_kernel<<<grid_size, block_size, shared_memory_size>>>(arr, output, N);

        hipDeviceSynchronize();  // Wait for the kernel to finish

        // Final reduction on the host
        GPUtype final_sum = 0.0;
        for (int i = 0; i < grid_size; i++) {
            final_sum += output[i];
        }

        // Cleanup
        hipFree(output);

        return final_sum;
    }

}