#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "rism3d_closure_kh.hpp"
#include <cmath>
using namespace std;

namespace rism3d_c {

    /////////////// KERNELS ///////////////
    __global__ void k_set_heaviside(GPUtype *heaviside, GPUtype *huv, int N) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < N) {
            // Assign 0 if huv[idx] > 0, otherwise assign 1
            heaviside[idx] = (huv[idx] > 0) ? 0 : 1;
        }
    }

    __global__ void k_guv(GPUtype *uuv, GPUtype *guv, GPUtype *huv, GPUtype *cuv, int size){

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        GPUtype expoent;

        if(idx < size){
            expoent = -uuv[idx] + huv[idx] - cuv[idx];
            if(expoent >= 0){
                guv[idx] = 1 + expoent;
            }
            else{
                guv[idx] = exp(expoent);
            }
        }
    }

    __global__ void k_excessChemicalPotential_grid(GPUtype *heaviside, GPUtype *huv, GPUtype *cuv, 
                                                   GPUtype *mu_grid, int Nx, int Ny, int Nz,
                                                   GPUtype density, GPUtype voxelVolume){
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < Nx * Ny * Nz){
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            int id = idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz;

            mu_grid[gid] = (0.5 * huv[id] * huv[id] * heaviside[id] - 0.5 * huv[id] * cuv[id] - cuv[id]) * density * voxelVolume;
        }
    }

    __global__ void k_excessChemicalPotential_lr_grid(GPUtype *heaviside, GPUtype *huv, GPUtype *cuv, 
                                                      GPUtype *mu_grid, int Nx, int Ny, int Nz,
                                                      GPUtype density, GPUtype voxelVolume,
                                                      GPUtype solvent_charge, GPUtype* dcfLongRangeAsympR,
                                                      GPUtype solute_totalcharge, GPUtype solvent_charge_sp,
                                                      GPUtype* tcfLongRangeAsympR){
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < Nx * Ny * Nz){
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            int id = idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz;
            int id_nonpadded = idx * Ny * Nz + idy * Nz + idz;

            // mu_grid[gid] = ((0.5 * huv[id] * huv[id] * heaviside[id] - 0.5 * huv[id] * cuv[id] - cuv[id]) ) * density * voxelVolume;

            GPUtype cuvlr = solvent_charge * dcfLongRangeAsympR[id_nonpadded];
            GPUtype huvlr = solvent_charge_sp * tcfLongRangeAsympR[id_nonpadded];
            
            if(solute_totalcharge * solvent_charge_sp <= 0.0){
                mu_grid[gid] = ((0.5 * huv[id] * huv[id] * heaviside[id] - 0.5 * huv[id] * cuv[id] - cuv[id]) + 0.5 * cuvlr * huvlr) * density * voxelVolume;
            }
            else{
                mu_grid[gid] = ((0.5 * huv[id] * huv[id] * heaviside[id] - 0.5 * huv[id] * cuv[id] - cuv[id]) + 0.5 * huvlr * (cuvlr - huvlr)) * density * voxelVolume;
            }
            
        }
    }

    __global__ void k_solvPotEne_grid(GPUtype *guv, GPUtype *uuv, GPUtype *solvPotEne_grid, 
                                      int Nx, int Ny, int Nz,
                                      GPUtype density, GPUtype voxelVolume){
        int gid = blockIdx.x * blockDim.x + threadIdx.x;

        if(gid < Nx * Ny * Nz){
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            int id = idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz;

            solvPotEne_grid[gid] = (guv[id] * uuv[id]) * density * voxelVolume;
        }
    }

    __global__ void k_akirkwoodBuff(GPUtype* huv, GPUtype solvent_charge_sp, 
                                    GPUtype* tcfLongRangeAsympR, 
                                    int Nx, int Ny, int Nz,
                                    GPUtype* block_kb) {

        extern __shared__ GPUtype shared_data[];

        int idx = blockIdx.x * blockDim.x + threadIdx.x; 
        int idy = blockIdx.y * blockDim.y + threadIdx.y; 
        int idz = blockIdx.z * blockDim.z + threadIdx.z; 

        int tid = threadIdx.x * (blockDim.y * blockDim.z) + threadIdx.y * blockDim.z + threadIdx.z; // Flattened thread ID in the block

        int global_index = (idx * Ny * Nz) + (idy * Nz) + idz; // non-padded indexing
        int global_padded_index = (idx * Ny * (Nz + 2)) + (idy * (Nz + 2)) + idz; // padded indexing

        // Load elements into shared memory
        if (idx < Nx && idy < Ny && idz < Nz) {
            shared_data[global_index] = huv[global_padded_index] - solvent_charge_sp * tcfLongRangeAsympR[global_index];
        } else {
            shared_data[tid] = 0.0;
        }

        __syncthreads();  // Synchronize threads in the block

        // Perform parallel reduction in shared memory:
        // This step will sum pair of values while reduicing the
        // the number of elements to be computed. Ex.: for the first block
        // containing 8 values
        // arr = [0 1 2 3 4 5 6 7]
        // for stride = 1:
        //     tid = 0, 2, 4, 6
        //     arr = [1 1 5 3 9 5 13 7]
        // for stride = 2:
        //     tid = 0, 4
        //     arr = [6 1 5 3 22 5 13 7]
        // for stride = 4
        //     tid = 0
        //     arr = [28 1 5 3 22 5 13 7]
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            if (tid % (2 * stride) == 0) {
                shared_data[tid] += shared_data[tid + stride];
            }
            __syncthreads();  // Ensure all threads have updated shared memory
        }

        for (int stride = 1; stride < (blockDim.x * blockDim.y * blockDim.z); stride *= 2) {
            if (tid % (2 * stride) == 0 && (tid + stride) < (blockDim.x * blockDim.y * blockDim.z)) {
                shared_data[tid] += shared_data[tid + stride];
            }
            __syncthreads();
        }

        // Write the result of the reduction in the first thread of the block
        // The value of the first thread of each block (which stores the total
        // sum for that block) will be asigned to the output array at the block 
        // id position
        if (tid == 0) {
            int block_id = (blockIdx.x * gridDim.y * gridDim.z) + (blockIdx.y * gridDim.z) + blockIdx.z; // ✅ Row-major block index
            block_kb[block_id] = shared_data[0];
        }
    }

    __global__ void k_akirkwoodBuff2(GPUtype* huv, GPUtype solvent_charge_sp, 
                                     GPUtype* tcfLongRangeAsympR, 
                                     int Nx, int Ny, int Nz, GPUtype* block_kb) {

        extern __shared__ GPUtype shared_data[];

        int gid = threadIdx.x + blockIdx.x * blockDim.x; // global index
        int tid = threadIdx.x; // thread's local id within the block (ranging from 0 to blockDim.x - 1).

        // Load elements into shared memory
        if (gid < Nx * Ny * Nz) {
            int idx = gid / (Ny * Nz);
            int idy = (gid / Nz) % Ny;
            int idz = gid % Nz;

            shared_data[tid] = huv[idx * Ny * (Nz + 2) + idy * (Nz + 2) + idz] - solvent_charge_sp * tcfLongRangeAsympR[gid];
        } else {
            shared_data[tid] = 0.0;
        }

        __syncthreads();  // Synchronize threads in the block

        // Perform parallel reduction in shared memory:
        // This step will sum pair of values while reduicing the
        // the number of elements to be computed. Ex.: for the first block
        // containing 8 values
        // arr = [0 1 2 3 4 5 6 7]
        // for stride = 1:
        //     tid = 0, 2, 4, 6
        //     arr = [1 1 5 3 9 5 13 7]
        // for stride = 2:
        //     tid = 0, 4
        //     arr = [6 1 5 3 22 5 13 7]
        // for stride = 4
        //     tid = 0
        //     arr = [28 1 5 3 22 5 13 7]
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            if (tid % (2 * stride) == 0) {
                shared_data[tid] += shared_data[tid + stride];
            }
            __syncthreads();  // Ensure all threads have updated shared memory
        }

        // Write the result of the reduction in the first thread of the block
        // The value of the first thread of each block (which stores the total
        // sum for that block) will be asigned to the output array at the block 
        // id position
        if (tid == 0) {
            block_kb[blockIdx.x] = shared_data[0];
        }
    }

    /////////////// KERNEL WRAPPER FUNCTIONS ///////////////

    void kh :: cu_guv(GPUtype *uuv, GPUtype *guv, GPUtype *huv, GPUtype *cuv, int size){
        int num_blocks = (size + 255) / 256;
        int num_threads = 256;

        k_guv<<<num_blocks, num_threads>>>(uuv, guv, huv, cuv, size);
        hipDeviceSynchronize();
    }

    void kh :: cu_set_heaviside(GPUtype *heaviside, GPUtype *huv, int N){
        int num_threads = 256;
        int num_blocks = (N + num_threads - 1) / num_threads;

        k_set_heaviside<<<num_blocks, num_threads>>>(heaviside, huv, N);
    }

    void kh :: cu_excessChemicalPotential_grid(GPUtype *heaviside, GPUtype *huv, GPUtype *cuv, 
                                               GPUtype *mu_grid, int Nx, int Ny, int Nz,
                                               GPUtype density, GPUtype voxelVolume){
        int num_threads = 256;
        int num_blocks = ((Nx * Ny * Nz) + num_threads - 1) / num_threads;

        k_excessChemicalPotential_grid<<<num_blocks, num_threads>>>(heaviside, huv, cuv, 
                                                                    mu_grid, Nx, Ny, Nz,
                                                                    density, voxelVolume);
    }

    void kh :: cu_excessChemicalPotential_lr_grid(GPUtype *heaviside, GPUtype *huv, GPUtype *cuv, 
                                                  GPUtype *mu_grid, int Nx, int Ny, int Nz,
                                                  GPUtype density, GPUtype voxelVolume,
                                                  GPUtype solvent_charge, GPUtype* dcfLongRangeAsympR,
                                                  GPUtype solute_totalcharge, GPUtype solvent_charge_sp,
                                                  GPUtype* tcfLongRangeAsympR){
        int num_threads = 256;
        int num_blocks = ((Nx * Ny * Nz) + num_threads - 1) / num_threads;

        k_excessChemicalPotential_lr_grid<<<num_blocks, num_threads>>>(heaviside, huv, cuv, 
                                                                       mu_grid, Nx, Ny, Nz,
                                                                       density, voxelVolume,
                                                                       solvent_charge, dcfLongRangeAsympR,
                                                                       solute_totalcharge, solvent_charge_sp,
                                                                       tcfLongRangeAsympR);
    }

    void kh :: cu_solvPotEne_grid(GPUtype *guv, GPUtype *uuv, GPUtype *solvPotEne_grid, 
                                  int Nx, int Ny, int Nz,
                                  GPUtype density, GPUtype voxelVolume){
        int num_threads = 256;
        int num_blocks = ((Nx * Ny * Nz) + num_threads - 1) / num_threads;

        k_solvPotEne_grid<<<num_blocks, num_threads>>>(guv, uuv, solvPotEne_grid,
                                                       Nx, Ny, Nz,
                                                       density, voxelVolume);
    }

    GPUtype kh :: cu_akirkwoodBuff(GPUtype* huv, GPUtype solvent_charge_sp, 
                                   GPUtype* tcfLongRangeAsympR, 
                                   int Nx, int Ny, int Nz){

        int N = Nx * Ny * Nz;

        int block_size = 256; // number of threads in each block
        int grid_size = (N + block_size - 1) / block_size; // total number of blocks to be used
        int shared_memory_size = block_size * sizeof(GPUtype);

        GPUtype *block_kb;
        hipMallocManaged(&block_kb, (N + 255) / 256 * sizeof(GPUtype));

        k_akirkwoodBuff2<<<grid_size, block_size, shared_memory_size>>>(huv, solvent_charge_sp, 
                                                                        tcfLongRangeAsympR, 
                                                                        Nx, Ny, Nz, block_kb);

        hipDeviceSynchronize();

        GPUtype akb_iv = 0;
        for (int i = 0; i < grid_size; i++) {
            akb_iv += block_kb[i];
        }

        hipFree(block_kb);
        return akb_iv;
    }

}